//implement a simple reduction(sum) kernel
#include <iostream>
#include <hip/hip_runtime.h>

// Define constants
//#define N 9999
#define BLK_SIZE 128
#define COARSE 4

using namespace std;
//usaully we have to increase the rate at which our kernel gets the data.
//use mechanisms like memory coalescing and conflict-free access at first.


//each thread block is responsible for BLK_SIZE x COARSE chunk of input data.
__global__ void reduction_divergence(int *input, int *output, int len) {
    unsigned int segmentStart = blockIdx.x * BLK_SIZE * COARSE;
    //compute data index for each thread
    unsigned int tid  = threadIdx.x;

    __shared__ int smem[BLK_SIZE];
    smem[tid] = 0;
    __syncthreads();

    //reduce the smem size of blocksize.
    //load 128 consecutive bytes and so on.
    //check if it is 4-sector returns per loop. //how to check?
    for (int i = 0; i<COARSE; i++) {
      int iidx   = segmentStart + (BLK_SIZE * i) + tid;
      if (iidx < len)
        smem[tid] += input[iidx];
    }
    __syncthreads();

    //since we have smem of block_size, start with block_size / 2 stride
    if (tid < 64) smem[tid] += smem[tid + 64];
    __syncthreads();

    //wrap reduce unroll
    //volatile required to prevent re-ordering by compiler.
    volatile int *vsem = smem;
    if (tid < 32) vsem[tid] += vsem[tid + 32];
    __syncthreads();

    if (tid < 16) vsem[tid] += vsem[tid + 16];
    if (tid < 8)  vsem[tid] += vsem[tid + 8];
    if (tid < 4)  vsem[tid] += vsem[tid + 4];
    if (tid < 2)  vsem[tid] += vsem[tid + 2];
    if (tid < 1)  vsem[tid] += vsem[tid + 1];
  
    if (tid == 0) {
      output[blockIdx.x] = vsem[tid];
    }
    return;
}

int cpu_reduction_func(int *input, int len) {
  int sum = 0.0f;
  for (int i=0; i<len; i++) {
    sum += input[i];
  }
  return sum;
}

int main(int argc, char **argv) {
    int N = atoi(argv[1]);
    //cout << N << endl;
    // Host and device pointers
    int *h_A, *h_C;
    int *d_A, *d_C;

    // Allocate memory on host
    h_A = (int*)malloc(N * sizeof(int));

    // Initialize input data
    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<int>(i);
    }

    // Allocate memory on device
    hipMalloc((void**)&d_A, N * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, N * sizeof(int), hipMemcpyHostToDevice);

    // <<< Launch kernel here when implemented >>>
    size_t ngrids = (N+COARSE*BLK_SIZE-1)/(COARSE*BLK_SIZE), nblocks = BLK_SIZE;

    h_C = (int *)malloc(ngrids * sizeof(int));
    hipMalloc((void**)&d_C, ngrids * sizeof(int));

    hipEvent_t start, stop;
    float milliseconds = 0;
    
    // Create CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Record the start event
    hipEventRecord(start);
    
    // Launch the kernel
    reduction_divergence<<<ngrids, nblocks>>>(d_A, d_C, N);
    hipDeviceSynchronize();

    // Record the stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel Execution Time: " << milliseconds << " ms" << std::endl;


    // Copy results back to host
    hipMemcpy(h_C, d_C, ngrids * sizeof(int), hipMemcpyDeviceToHost);

    int result = cpu_reduction_func(h_A, N);
    int gpu_result = cpu_reduction_func(h_C, ngrids);

    if (abs(result - gpu_result) > 1e-3) {
      std::cout << "\n Actual : " << gpu_result << " != " << " Expected : " << result << "\n";
      std::exit(-1);
    }

    std::cout << "\n reduction complete \n";
    // Cleanup
    hipFree(d_A);
    hipFree(d_C);
    free(h_A);
    free(h_C);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}