//implement a simple reduction(sum) kernel
#include <iostream>
#include <hip/hip_runtime.h>

// Define constants
//#define N 9999
#define BLK_SIZE 128

using namespace std;
//each thread block is responsible for BLK_SIZE x 2 chunk of input data.
__global__ void reduction_divergence(int *input, int *output, int len) {
    unsigned int segmentStart = blockIdx.x * BLK_SIZE * 2;
    //compute data index for each thread
    unsigned int tid  = threadIdx.x;

    __shared__ int smem[BLK_SIZE*2];
  
    smem[tid]            = segmentStart + tid < len ? input[segmentStart + tid] : 0;
    smem[tid+blockDim.x] = segmentStart + blockDim.x + tid < len ? input[segmentStart + blockDim.x + tid] : 0;

    __syncthreads();

    for (int stride = blockDim.x; stride > 0; stride /= 2) {
      if (tid < stride) {
        smem[tid] += smem[tid+stride];
      }
      __syncthreads();
    }

    if (tid == 0) {
      output[blockIdx.x] = smem[tid];
    }
    return;
}

int cpu_reduction_func(int *input, int len) {
  int sum = 0.0f;
  for (int i=0; i<len; i++) {
    sum += input[i];
  }
  return sum;
}

int main(int argc, char **argv) {
    int N = atoi(argv[1]);
    //cout << N << endl;
    // Host and device pointers
    int *h_A, *h_C;
    int *d_A, *d_C;

    // Allocate memory on host
    h_A = (int*)malloc(N * sizeof(int));

    // Initialize input data
    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<int>(i);
    }

    // Allocate memory on device
    hipMalloc((void**)&d_A, N * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, N * sizeof(int), hipMemcpyHostToDevice);

    // <<< Launch kernel here when implemented >>>
    size_t ngrids = (N+2*BLK_SIZE-1)/(2*BLK_SIZE), nblocks = BLK_SIZE;

    h_C = (int *)malloc(ngrids * sizeof(int));
    hipMalloc((void**)&d_C, ngrids * sizeof(int));

    reduction_divergence<<<ngrids, nblocks>>>(d_A, d_C, N);
    hipDeviceSynchronize();

    // Copy results back to host
    hipMemcpy(h_C, d_C, ngrids * sizeof(int), hipMemcpyDeviceToHost);

    int result = cpu_reduction_func(h_A, N);
    int gpu_result = cpu_reduction_func(h_C, ngrids);

    if (abs(result - gpu_result) > 1e-3) {
      std::cout << "\n Actual : " << gpu_result << " != " << " Expected : " << result << "\n";
      std::exit(-1);
    }

    std::cout << "\n reduction complete \n";
    // Cleanup
    hipFree(d_A);
    hipFree(d_C);
    free(h_A);
    free(h_C);
    return 0;
}