#include<hip/hip_runtime.h>
#include<mma.h>

#include<iostream>
#include<random>
#include<utility>

#define N 32
#define M 32
#define K 32

using namespace std;
using namespace nvcuda;

template<typename T>
struct Matrix {
  Matrix(int rows, int cols) {
    this->rows = rows;
    this->cols = cols;
    ptr = (T *)malloc(this->rows * this->cols * sizeof(T));
  }
  ~Matrix() {
    if (ptr)
      free(ptr);
  }
  void fill() {
    std::default_random_engine random_engine(0);
    uniform_int_distribution<T> uniform_dist(-128, 127);
    for (int i=0; i<(rows * cols); i++)
      ptr[i] = uniform_dist(random_engine);
  }
  size_t getSize() {
    return rows * cols * sizeof(T);
  }
  static Matrix createMatrix(int rows, int cols) {
    return Matrix(rows, cols);
  }

  T *ptr;
  size_t rows, cols;
};


template<typename T1, typename T2>
__global__ void mm(T1 *a, T1 *b, T2 *c, int m, int k, int n) {

  const int num_wraps_x = 4, num_wraps_y = 4;
  const int wrap_m = 16, wrap_n = 16, wrap_k = 16;

  __shared__ T1 matA[num_wraps_y * wrap_m][256];
  __shared__ T1 matB[256][num_wraps_x * wrap_n];

  //do coaleased memory access.
  //start row in A of this block.
  int blk_row_a = (num_wraps_y * wrap_m) * blockIdx.y;
  int cr_size = blockDim.y, cc_size = blockDim.x;

  for (int chunk_row=0; chunk_row < (blockDim.y * wrap_m); chunk_row += cr_size) {
    int sm_row = chunk_row + threadIdx.y;
    int gm_row = blk_row_a + sm_row;

    //for matrix A this block will do from blk_row_a to blk_row_a + (num_wrap_y * wrap_m)
    //this means k is divisible by blockDim.x
    for (int chunk_col=0; chunk_col < k; chunk_col += cc_size) {
      int sm_col = chunk_col + threadIdx.x;

      int a_linear_addr = gm_row * k + sm_col;
      matA[sm_row][sm_col] = a[a_linear_addr];
    }
  }

  //start col in B of this block.
  //(4x128) threads here.
  int blk_col_b = (num_wraps_x * wrap_n) * blockIdx.x;
  cc_size = num_wraps_x, cr_size = blockDim.x;
  for (int chunk_col=0; chunk_col < (num_wraps_x * wrap_n); chunk_col += cc_size) {//maps to thread y dim
    int sm_col = chunk_col + threadIdx.y;
    int gm_col = blk_col_b + sm_col;

    for (int chunk_row=0; chunk_row < k; chunk_row += cr_size) {
      int sm_row = chunk_row + threadIdx.x;

      int b_linear_addr = sm_row * n + gm_col;
      matB[sm_row][sm_col] = b[b_linear_addr]; 
    }
  }

  __syncthreads();
  //mat(64, k) and mat(k, 64)
  const int wrap_size = 32;

  int lda = 256, ldb = num_wraps_x * wrap_n;
  int local_wrap_x = threadIdx.x / wrap_size;
  int local_wrap_y = threadIdx.y;

  //each wrap is responsible for 16 x 16 chunk for output matrix.
  //a_frag
  wmma::fragment<wmma::matrix_a, wrap_m, wrap_n, wrap_k, T1, wmma::row_major> a_frag;
  //b_frag
  wmma::fragment<wmma::matrix_b, wrap_m, wrap_n, wrap_k, T1, wmma::row_major> b_frag;
  //accum_frag
  wmma::fragment<wmma::accumulator, wrap_m, wrap_n, wrap_k, T2> accum_frag;

  wmma::fill_fragment(accum_frag, 0);

  for (int i=0; i<k; i+=wrap_k) {
    //a smem matrix.
    int a_wrap_row = local_wrap_y * wrap_m, a_wrap_col = i;
    //b smem matrix.
    int b_wrap_row = i, b_wrap_col = local_wrap_x * wrap_n;

    T1 *a_sm_ptr = &matA[a_wrap_row][a_wrap_col];
    T1 *b_sm_ptr = &matB[b_wrap_row][b_wrap_col];

    //load
    wmma::load_matrix_sync(a_frag, a_sm_ptr, lda);
    wmma::load_matrix_sync(b_frag, b_sm_ptr, ldb);

    wmma::mma_sync(accum_frag, a_frag, b_frag, accum_frag);
  }

  //store
  int wrap_x = (blockDim.x * blockIdx.x + threadIdx.x) / wrap_size;
  int wrap_y = (blockDim.y * blockIdx.y) + threadIdx.y;

  int c_start_row = wrap_y * 16, c_start_col = wrap_x * 16;
  T2 *c_ptr = c + (c_start_row * n + c_start_col);
  wmma::store_matrix_sync(c_ptr, accum_frag, n, wmma::mem_row_major);
  return;
}

template<typename T1, typename T2>
Matrix<T2> launch_matmul_kernel(Matrix<T1>& A, Matrix<T1>& B) {
  //device matrix allocate
  T1 *dmatrix_a, *dmatrix_b;
  T2 *dmatrix_c;

  Matrix<T2> C = Matrix<T2>::createMatrix(A.rows, B.cols);

  hipMalloc((void **)&dmatrix_a, A.getSize());
  hipMalloc((void **)&dmatrix_b, B.getSize());
  hipMalloc((void **)&dmatrix_c, C.getSize());

  //device matrix copy
  hipMemcpy(dmatrix_a, A.ptr, A.getSize(), hipMemcpyHostToDevice);
  hipMemcpy(dmatrix_b, B.ptr, B.getSize(), hipMemcpyHostToDevice);

  //kernel stepup
  //each block has (4 x 4 wraps)
  const int num_wraps_x = 4;//horizontal
  const int num_wraps_y = 4;//vertical
  const int wrap_m = 16;
  const int wrap_n = 16;
  int n = C.cols, m = C.rows;
  dim3 block, grid;
  block.x = num_wraps_x * 32;
  block.y = num_wraps_y;

  //output wil be of size (m * n)
  //map horizontal blocks to columns
  //vertical blocks to rows
  grid.x  = (n + num_wraps_x * wrap_n - 1) / (num_wraps_x * wrap_n);
  grid.y  = (m + num_wraps_y * wrap_m - 1) / (num_wraps_y * wrap_m);

  std::cout << "\n grid  dim : " << grid.x <<  "," << grid.y;
  std::cout << "\n block dim : " << block.x << "," << block.y;
  
  hipFuncSetAttribute(reinterpret_cast<const void*>(mm<int8_t, int>), 
    hipFuncAttributeMaxDynamicSharedMemorySize, 65536);

  mm<int8_t, int><<<grid, block>>>(dmatrix_a, dmatrix_b, dmatrix_c, A.rows, A.cols, B.cols);
  hipDeviceSynchronize();
  //copy result
  hipMemcpy(C.ptr, dmatrix_c, C.getSize(), hipMemcpyDeviceToHost);
  return C;
}

template<typename T1, typename T2>
Matrix<T2> matrixMultiply(Matrix<T1> &A, Matrix<T1> &B) {
  Matrix<T2> C = Matrix<T2>::createMatrix(A.rows, B.cols);
  
  int m = A.rows, k = A.cols, n = B.cols;
  for (int i=0; i<m; i++) {
    for (int j=0; j<n; j++) {
      T2 sum = 0;
      for (int z=0; z<k; z++) {
        sum += A.ptr[i * k + z] * B.ptr[z * n + j];
      }
      C.ptr[i * n + j] = sum;
    }
  }
  return C;
}

template<typename T>
bool verify(Matrix<T> &gpu_c, Matrix<T> &cpu_c) {
  if ((gpu_c.rows != cpu_c.rows) || (gpu_c.cols != cpu_c.cols)) {
    std::cout << "\n shape mismatch";
    return false;
  }

  for (int i=0; i<cpu_c.rows; i++) {
    for (int j=0; j<cpu_c.cols; j++) {
      if (gpu_c.ptr[i * cpu_c.cols + j] != cpu_c.ptr[i * cpu_c.cols + j])
        return false;
    }
  }
  return true;
}

//Matrix<int8_t>(rows, cols);
int main(int argc, char** argv) {
  if (argc < 4) {
    return 0;
  }
  int m = atoi(argv[1]), k = atoi(argv[2]), n = atoi(argv[3]);

  using T1 = int8_t;
  using T2 = int;
  //input allocate
  Matrix A = Matrix<T1>::createMatrix(m, k);
  Matrix B = Matrix<T1>::createMatrix(k, n);
  
  //fill input matrices
  A.fill();
  B.fill();

  auto mma_c = launch_matmul_kernel<T1, T2>(A, B);

  auto cpu_c = matrixMultiply<T1, T2>(A, B);

  if(!verify(mma_c, cpu_c)) {
    std::cout << "\n mismatch failed";
  } else {
    std::cout << "\n kernel passed";
  }
  return 0;
}